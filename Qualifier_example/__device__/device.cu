#include <stdio.h>
#include <hip/hip_runtime.h>


#define ELEMENTS  1000
#define THREADS_PER_BLOCK  8

hipEvent_t start_time = nullptr;
hipEvent_t end_time = nullptr;
float elapsed_time = 0.0f;

// 1. __constant__ memmory: bo nho hang so chi doc, danh cho moi thread (cap phat boi CPU)
__constant__ int constMultiplier; //uninitialized of Data (BSS)

// 2. __device__: chi thi cho biet ham nay chay tren GPU va chi GPU goi duoc (ham con GPU)
__device__ int square(int x){
  return x * x;
}

//Cac ham nay mac dinh chay o CPU 
__host__ void startTimer(void){
  hipEventCreate(&start_time);
  hipEventCreate(&end_time);
  hipEventRecord(start_time, 0);
}

__host__ float elapsedTimer(void){
  hipEventRecord(end_time, 0);
  hipEventSynchronize(end_time); //Dam bao kernel da chay xong

  hipEventElapsedTime(&elapsed_time, start_time, end_time);
  hipEventDestroy(start_time);
  hipEventDestroy(end_time);

  return elapsed_time;
}

// 3. __global__: Ham goi tu CPU nhung thuc thi tren GPU 
__global__ void processArray(int *arr){
  //4. __shared__ memory: Bo nho chia se giua cac thread trong cung 1 block
  __shared__ int temp[THREADS_PER_BLOCK];

  /**
   * @brief Tinh chi so toan cuc (global index) cua thread trong toan bo luoi (grid)
   * @param threadIdx - Chi so cua thread trong block (vd: tu 0 den 255 neu co 256 thread trong block)
   * @param blockIdx - Chi so cua block trong grid (vd: tu 0 den 3 neu co 4 blocks trong 1 grid)
   * @param blockDim - So thread trong moi block
   */
  //Idx: vi tri thread trong toan bo chuong trinh 
  int idx = threadIdx.x + blockIdx.x * blockDim.x;

  if(idx < ELEMENTS){
    int val = arr[idx];

    //Goi ham device va nhan voi hang so constant 
    temp[threadIdx.x] = square(val) * constMultiplier; //Gan cho moi thread trong block 

    //Dong bo cac thread trong block truoc khi gui ket qua ve 
    __syncthreads();

    arr[idx] = temp[threadIdx.x];
  }
}

int main(void){
  int count_elements = 0;
  int host_arr[ELEMENTS];
  int *device_arr;

  //Khoi tao mang 
  for(int i = 0; i < ELEMENTS; i++){
    host_arr[i] = i;
  }

  //Cap phat bo nho tren GPU 
  hipMalloc(&device_arr, sizeof(int) * ELEMENTS);
  hipMemcpy(device_arr, host_arr, sizeof(int) * ELEMENTS, hipMemcpyHostToDevice);

  //Sao chep hang so vao __constant__ memory
  int multiplier = 2;
  hipMemcpyToSymbol(HIP_SYMBOL(constMultiplier), &multiplier, sizeof(int));

  //Tinh so block va goi kernel 
  int numBlocks = (ELEMENTS + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

  startTimer();
  processArray <<< numBlocks, THREADS_PER_BLOCK >>>(device_arr);
  elapsedTimer();

  //Sao chep ket qua ve lai CPU 
  hipMemcpy(host_arr, device_arr, sizeof(int) * ELEMENTS, hipMemcpyDeviceToHost);

   printf("Ket qua sau khi thuc thi tren GPU: \n");
  for(int i = 0; i < ELEMENTS; i++){
    printf("%d ", host_arr[i]);
    count_elements++;
    if(count_elements == 20){
      count_elements = 0;
      printf("\n");
    }
  }
  printf("\nThoi gian thuc thi tren kernel: %.6fms", elapsed_time);
  hipFree(device_arr);
  return 0;
}