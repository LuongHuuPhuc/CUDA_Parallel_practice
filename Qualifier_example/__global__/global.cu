#include <iostream>
#include <hip/hip_runtime.h>


#define N   1000 //Kich thuoc mang
/**
 * @note Chi thi __global__ duoc gan cho cac ham kernel, nghia la: 
 * \note - Ham duoc goi tu CPU nhung lai chay tren GPU
 * \note - Nen dung chi thi nay cho cac ham chinh khoi chay tren GPU (kernel)
 * \note - Khong phu hop cho cac ham tinh toan nho/modular
 * \note - Khong duoc phep tra ve gia tri
 */
__global__ void myKernel(int *arg){
  int idx =  threadIdx.x + blockIdx.x * blockDim.x;
  if(idx < N){
    arg[idx] += 1;
  }
}

int main(void){
  int count_elements = 0;
  int host_a[N]; //Mang tren CPU (host)
  int *device_a; //Con tro tren GPU (device)

  //Khoi tao mang host_a
  for(int i = 0; i < N; i++){
    host_a[i] = i;
  }

  //Cap phat bo nho tren GPU
  hipMalloc((void**)&device_a, sizeof(int) * N);

  //Sao chep du lieu tu CPU sang GPU
  hipMemcpy(device_a, host_a, sizeof(int) * N, hipMemcpyHostToDevice);

  /**
   * @brief Goi kernel tren GPU voi n block, moi block m thread 
   * @note Muc tieu: Chia N phan tu thanh nhieu block, voi 1 thread dam nhiem 1 phan tu, sao cho:
   * \note - Tong so thread >= N
   * \note - Moi block co dung threadsPerBlock thread (ngoai tru block cuoi neu du)
   * Neu N la so khong chia het cho `threadsPerBlock` thi se xay ra truong hop bo sot nhung phan tu cuoi cung
   * => De giai quyet cho van de, thi su dung giai phap lam tron len (chia lay tran) de sinh ra thua threads, 
   * dam bao khong co phan tu nao bi bo sot
   */
  int threadsPerBlock = 9;    
  int numBlocks = (N + threadsPerBlock - 1) / threadsPerBlock; //Ceiling
  //Goi kernel tren CPU
  myKernel <<< numBlocks, threadsPerBlock >>> (device_a);

  //Dong bo hoa GPU 
  hipDeviceSynchronize();

  //Sao chep du lieu tro ve CPU 
  hipMemcpy(host_a, device_a, sizeof(int) * N, hipMemcpyDeviceToHost);

  //In ra ket qua 
  printf("Ket qua sau khi cong 1 tren GPU: \n");
  for(int i = 0; i < N; i++){
    printf("%d ", host_a[i]);
    count_elements++;
    if(count_elements == 20){
      count_elements = 0;
      printf("\n");
    }
  }

  //Giai phong bo nho tren GPU
  hipFree(device_a);

  return 0;
}