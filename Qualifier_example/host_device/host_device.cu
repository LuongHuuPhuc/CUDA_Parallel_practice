#include <stdio.h>
#include <hip/hip_runtime.h>


#define THREADS_PER_BLOCK 1
#define NUMBLOCKS 1

//Ham chay duoc ca tren GPU va CPU
__host__ __device__ int add(int a, int b){
  return a + b;
}

__global__ void addKernel(int *device_result, int a, int b){
  *device_result = add(a, b); //Goi tren GPU
}

int main(void){
  int a = 10, b = 20;

  int cpu_result = add(a, b); //Goi tren CPU
  printf("Result call on CPU: %d\n", cpu_result);

  //Neu khai bao la int device_result thi day chi la bien thuong, neu ham can truyen con tro vao thi phai lay dia chi cua bien do: &device_result
  int *device_result; //*device_result (dereference pointer): Gia tri cua con tro tai dia chi do
  int host_result;
  hipMalloc((void**)&device_result, sizeof(int));
  addKernel <<< NUMBLOCKS, THREADS_PER_BLOCK >>> (device_result, a, b); //Truyen vao con tro device_result
  hipMemcpy(&host_result, device_result, sizeof(int), hipMemcpyDeviceToHost);
  printf("Result call on GPU: %d\n", host_result);

  hipFree(device_result);
  return 0;
}