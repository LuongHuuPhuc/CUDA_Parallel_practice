#include <stdio.h>
#include <hip/hip_runtime.h>

#define N  18

__constant__ int factor; //Bien constant tren device

//Kernel nhan moi phan voi constant 
__global__ void multiplyConstant(int *out, int *in){
  int idx = threadIdx.x;
  if(idx < N){
    out[idx] = in[idx] * factor;
  }
}

int main(void){
  int h_in[N], h_out[N];
  int *d_in, *d_out;
  int h_factor = 5;
  hipEvent_t start_time = nullptr, end_time = nullptr; 
  float elapsed_time = 0.0f;

  //Khoi tao du lieu dau vao
  for(int i = 0; i < N; i++){
    h_in[i] =  i + 1;
  }
  
  //Cap phat bo nho tren device
  hipMalloc((void**)&d_in, sizeof(int) * N);
  hipMalloc((void**)&d_out, sizeof(int) * N);

  //Sao chep du lieu tu host sang device 
  hipMemcpy(d_in, h_in, sizeof(int) * N, hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(factor), &h_factor, sizeof(int)); //Gan gia tri cho bien constant

  hipEventCreate(&start_time);
  hipEventCreate(&end_time);
  hipEventRecord(start_time, 0);

  //Goi kernel    
  multiplyConstant <<< 1, N >>> (d_out, d_in);
  
  hipEventRecord(end_time, 0);
  hipEventSynchronize(end_time);
  hipEventElapsedTime(&elapsed_time, start_time, end_time);
  hipEventDestroy(start_time);
  hipEventDestroy(end_time);

  //Sao chep ket qua ve host 
  hipMemcpy(h_out, d_out, sizeof(int) * N, hipMemcpyDeviceToHost);

  printf("Ket qua nhan voi h_factor = %d:\n", h_factor);
  for(int i = 0; i < N; i++){
    printf("%d * %d = %d\n", h_in[i], h_factor, h_out[i]);
  }
  printf("Thoi gian thuc thi tren Kernel: %.10f\n", elapsed_time);

  hipFree(d_in);
  hipFree(d_out);
  return 0;
}
