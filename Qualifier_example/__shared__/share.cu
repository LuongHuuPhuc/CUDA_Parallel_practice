#include <stdio.h>
#include <hip/hip_runtime.h>

//__shared__: bo nho chia se giua cac thread trong 1 block

__global__ void addShared(int *out, int *in1, int *in2){
  __shared__ int temp1[256], temp2[256];

  int t_id = threadIdx.x; //So luong thread se duoc quyet dinh boi luc goi kernel <<< >>>

  temp1[t_id] = in1[t_id];
  temp2[t_id] = in2[t_id];

}