#include <iostream>
#include <hip/hip_runtime.h>


#define N (500 * 1000 * 1000) //1e9

//Thuv hien tren Kernel
__global__ void vectorAddGPU(const float *A, const float *B, float *C, int n){
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if(i < n){
    C[i] = A[i] + B[i];
  }
}

int main(void){
  float *A, *B, *C;
  float *d_A, *d_B, *d_C;
  
  A = new float[N]; //Mang dong A chua N phan tu 
  B = new float[N];
  C = new float[N];

  for(int i = 0; i < N; i++){
    A[i] = 1.0f;
    B[i] = 2.0f;
  }

  hipMalloc(&d_A, N * sizeof(float));
  hipMalloc(&d_B, N * sizeof(float));
  hipMalloc(&d_C, N * sizeof(float));

  hipMemcpy(d_A, A, sizeof(float) * N, hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, sizeof(float) * N, hipMemcpyHostToDevice);

  int blockSize = 256;
  int numBlocks = (N + blockSize - 1) / blockSize;
  
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);

  //Call ham kernel de thuc thi tren GPU
  vectorAddGPU<<<numBlocks, blockSize>>>(d_A, d_B, d_C, N);
  hipEventRecord(stop);
  hipMemcpy(d_C, C, sizeof(float), hipMemcpyHostToDevice);

  hipEventSynchronize(stop);
  float milliseconds = 0.0f;
  hipEventElapsedTime(&milliseconds, start, stop);

  std::cout << "GPU time: " << milliseconds / 1000.0f << " seconds\n";

  delete[] A;
  delete[] B;
  delete[] C;

  hipFree(d_A);
  hipFree(d_B);  
  hipFree(d_C);

  return 0;
}