#include <iostream>
#include <math.h>
#include <hip/hip_runtime.h>


__global__ void add(int n, float *x, float *y){
  int index = blockDim.x * blockIdx.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for(int i = index; i < n; i += stride){
    y[i] = x[i] + y[i];
  }
}

int main(void){
  int N = 1 << 20;
  float *x, *y;

  //Allocate Unified Memory (Bo nho thong nhat) -- Bo nho co tren truy cap duoc ca tu CPU va GPU 
  hipMallocManaged(&x, N * sizeof(float));
  hipMallocManaged(&y, N * sizeof(float));

  //Khoi tao mang 
  for(int i = 0; i < N; i++){
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  //Chay kernel voi 1 trieu phan tu tren GPU 
  int threadsPerBlock = 256;
  int numBlocks = (N + threadsPerBlock - 1) / threadsPerBlock;
  add <<< numBlocks, threadsPerBlock >>> (N, x, y);

  //Doi cho GPU hoan thanh truoc khi access vao host (CPU)
  hipDeviceSynchronize();

  //Check loi (Tat ca gia tri dung phai la 3.0f)
  float maxErr = 0.0f;
  for(int i = 0; i < N; i++){
    maxErr = fmax(maxErr, fabs(y[i] - 3.0f));
  }
  std::cout << "Max error: " << maxErr << std::endl;

  hipFree(x);
  hipFree(y);
  return 0;
}