#include <iostream>
#include <hip/hip_runtime.h>


/**
 * \note The nen can ca bien dich cho C++ cho CPU
 * \note nvcc (NVIDIA CUDA Compiler driver) khong phai la compiler C++ thuan
 * No chi la trinh dieu phoi (driver) de giup phan tach phan code GPU va code CPU, 
 * chuyen cac ham __global__, __device__,... thanh PTX (Parallel Thread Execution) 
 * hoac SASS (native Assembly cho GPU)
 * \note Phan CPU se duoc compile bang cl.exe/gcc (Vi co ban chuong trinh van nam tren CPU)
 * \note Sau do link 2 phan lai voi nhau
 */

//__global__ = chay tren GPU, goi tu CPU (host)
__global__ void hello_from_gpu(void){
  printf("Hello from GPU, thread: %d\n", threadIdx.x);
}

int main(void){
  hello_from_gpu<<<1, 10>>>(); //1 blocks, 10 threads

  hipDeviceSynchronize();

  std::cout << "Hello from CPU !" << std::endl;
  return 0;
}