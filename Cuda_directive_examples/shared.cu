

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void addShared(int* out) {
    __shared__ int temp[256];
    int idx = threadIdx.x;
    temp[idx] = idx;
    __syncthreads();
    out[idx] = temp[idx] + 1;
}

int main() {
    const int ARRAY_SIZE = 5;
    const int ARRAY_BYTES = ARRAY_SIZE * sizeof(int);

    int h_out[ARRAY_SIZE];
    int* d_out;
    hipMalloc((void**)&d_out, ARRAY_BYTES);

    addShared<<<1, ARRAY_SIZE>>>(d_out);

    hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);

    for (int i = 0; i < ARRAY_SIZE; i++) {
        printf("%d ", h_out[i]);
    }
    printf("\n");

    hipFree(d_out);
    return 0;
}
