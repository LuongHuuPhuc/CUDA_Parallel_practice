

#include <hip/hip_runtime.h>
#include <stdio.h>

__constant__ int factor;

__global__ void multiplyConstant(int* out, int* in) {
    int idx = threadIdx.x;
    out[idx] = in[idx] * factor;
}

int main() {
    const int ARRAY_SIZE = 5;
    const int ARRAY_BYTES = ARRAY_SIZE * sizeof(int);

    int h_in[ARRAY_SIZE] = {1, 2, 3, 4, 5};
    int h_out[ARRAY_SIZE];
    int h_factor = 3;

    int *d_in, *d_out;
    hipMalloc((void**)&d_in, ARRAY_BYTES);
    hipMalloc((void**)&d_out, ARRAY_BYTES);

    hipMemcpyToSymbol(HIP_SYMBOL(factor), &h_factor, sizeof(int));
    hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice);

    multiplyConstant<<<1, ARRAY_SIZE>>>(d_out, d_in);

    hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);

    for (int i = 0; i < ARRAY_SIZE; i++) {
        printf("%d ", h_out[i]);
    }
    printf("\n");

    hipFree(d_in);
    hipFree(d_out);
    return 0;
}
