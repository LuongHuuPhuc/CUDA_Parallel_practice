

#include <hip/hip_runtime.h>
#include <stdio.h>

__managed__ int data[5];

__global__ void init() {
    int idx = threadIdx.x;
    data[idx] = idx * 2;
}

int main() {
    init<<<1, 5>>>();
    hipDeviceSynchronize();

    for (int i = 0; i < 5; i++) {
        printf("%d ", data[i]);
    }
    printf("\n");

    return 0;
}
