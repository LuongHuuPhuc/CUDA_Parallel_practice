

#include <hip/hip_runtime.h>
#include <stdio.h>

__device__ float cube(float x) {
    return x * x * x;
}

__global__ void applyCube(float* d_out, float* d_in) {
    int idx = threadIdx.x;
    d_out[idx] = cube(d_in[idx]);
}

int main() {
    const int ARRAY_SIZE = 5;
    const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);

    float h_in[ARRAY_SIZE] = {1, 2, 3, 4, 5};
    float h_out[ARRAY_SIZE];

    float *d_in, *d_out;
    hipMalloc((void**)&d_in, ARRAY_BYTES);
    hipMalloc((void**)&d_out, ARRAY_BYTES);

    hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice);

    applyCube<<<1, ARRAY_SIZE>>>(d_out, d_in);

    hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);

    for (int i = 0; i < ARRAY_SIZE; i++) {
        printf("%f ", h_out[i]);
    }
    printf("\n");

    hipFree(d_in);
    hipFree(d_out);
    return 0;
}
